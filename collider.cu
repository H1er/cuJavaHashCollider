
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


char * getstr(FILE* f, char * str)
{
    char l = 'A';
    size_t size = 20;
    int pos = 0;
    str = (char*) malloc(size);
    
    while(l != '\0' && l != '\n')
    {
        scanf("%c", &l);

        str[pos] = l;
        pos++;

         if(pos > size)
        {
            str = (char*) realloc(str, size*2);
        }
    }

    fflush(stdin);
    
    return (char *)realloc(str, pos);
}

__device__ __host__  int shc(char* ca) 
{
    int h = 0, tam=0;

    while(ca[tam]!='\0')
        tam++;
      
    for (int i = 0; i < tam-1; i++) 
    {
        h = 31 * h + ca[i];
    }

    return h;
}

__device__ char * id2str(unsigned long long int n) 
{
    //Convert the id of the thread into the string in order to be checked
    int b=95, i, r, digit, p, count=0;
    char * a, * res;// clrscr();
    p=n;
 
    a = (char*) malloc(100); //cudaMallocManaged(&a, 100);

    do 
    {
        r=p%b;
        digit=32+r;
        a[count]=digit;
        count++;
        p=p/b;
    } 
    while(p!=0);

    res = (char*) malloc(count);

    for(i=count-1; i>=0; --i) res[count-i-1] = a[i];

    free(a);

    return res;
}

__global__ void findcollisions(int hash, int f)
{
    unsigned long long int id = (unsigned long long int) blockDim.x * (unsigned long long int) blockIdx.x + (unsigned long long int) threadIdx.x; //+ i* /*4e40*/;
    
    char* trystr = id2str(id);

    int hc = shc(trystr);

    printf("┤%s├ hashcode: %d\n", trystr, hc);
    
    if(hc == hash)
    {
        printf("Collision found with string %s, hash code: %d\n", trystr, hc);
    }
    else
    {
        int tam = 0;

        while(trystr[tam]!='\0')
            tam++;

        if(trystr[tam-1] == ' ')
        {
            
            
            char * reverse;
            reverse = (char *) malloc(tam);

            for(int i=tam-1; i>=0; --i) reverse[tam-i-1] = trystr[i];

            hc = shc(reverse);
            printf("┤%s├ hashcode: %d\n", reverse, hc);
            
            if(hc == hash)
                printf("Collision found with string %s, hash code: %d", trystr, hc);
            
    
            free(reverse);
        
        }
    }

    free(trystr);
   
}


int main(void)
{
    char* input_string= NULL;

    printf("Introduce una cadena: ");

   
    input_string = getstr(stdin, input_string);
    int hash = shc(input_string);
    
    printf("\nhashcode: %d\n", hash);

    findcollisions<<</*1073741824*/3,47>>>(hash, 0); //<<<2^23, 2^10>>>

   
    hipDeviceSynchronize();
    
    free(input_string);

    return 0;
}