
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


char * getstr(FILE* f, char * str)
{
    char l = 'A';
    size_t size = 20;
    int pos = 0;
    str = (char*) malloc(size);
    
    while(l != '\0' && l != '\n')
    {
        scanf("%c", &l);

        str[pos] = l;
        pos++;

         if(pos > size)
        {
            str = (char*) realloc(str, size*2);
        }
    }

    fflush(stdin);
    
    return (char *)realloc(str, pos);
}

__device__ __host__  int shc(char* ca) 
{
    int h = 0, tam=0;

    while(ca[tam]!='\0')
        tam++;
      
    for (int i = 0; i < tam-1; i++) 
    {
        h = 31 * h + ca[i];
    }

    return h;
}

__device__ char * id2str(unsigned long long int n) 
{
    //Convert the id of the thread into the string in order to be checked
    int b=95, i, r, digit, p, count=0;
    char * a, * res;// clrscr();
    p=n;
 
    a = (char*) malloc(100); //cudaMallocManaged(&a, 100);

    do 
    {
        r=p%b;
        digit=32+r;
        a[count]=digit;
        count++;
        p=p/b;
    } 
    while(p!=0);

    res = (char*) malloc(count+1);

    for(i=count-1; i>=0; --i) res[count-i-1] = a[i];

    res[count] = '\n';

    free(a);

    return res;
}

__global__ void findcollisions(int hash, int f)
{
    unsigned long long int id = (unsigned long long int) blockDim.x * (unsigned long long int) blockIdx.x + (unsigned long long int) threadIdx.x; //+ i* /*4e40*/;

    if(id % 1000000000 == 0)
    {
        printf("\nid = %d\n", id);
    }
    
    char *trystr = id2str(id);
    int hc=0;

    hc = shc(trystr);

    
    if(hc == hash)
    {
        printf("Collision found with hashcode %d for string  ┤%s├\n", hc, trystr);
    }
    else
    {
        int tam = 0;
        

        while(trystr[tam]!='\0')
            tam++;

        char c = '!';
        int n=0;

        for(int i=0;i<tam-1;i++)
        {
            if(c != trystr[i])
            {
                n = 1;
            }

        }

        if(n == 0 && tam >2)
        {
            printf("Tam = %d\n", tam-1);
        }

        if(trystr[tam-1] == ' ')
        {
            char * reverse;
            reverse = (char *) malloc(tam);

            for(int i=tam-1; i>=0; --i) reverse[tam-i-1] = trystr[i];

            hc = shc(reverse);
            
            if(hc == hash)
                printf("Collision found with hashcode %d for string ┤%s├", hc, trystr);
            
    
            free(reverse);
        }
    }
    free(trystr);
}


int main(void)
{
    char* input_string= NULL;

    printf("Introduce una cadena: ");

   
    input_string = getstr(stdin, input_string);
    int hash = shc(input_string);
    
    printf("hashcode: %d\n\n", hash);

    findcollisions<<<1073741824,1024>>>(hash, 0); //<<<2^23, 2^10>>>

   
    hipDeviceSynchronize();
    
    free(input_string);

    return 0;
}