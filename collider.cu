
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

char * getstr(FILE* f, char * str)
{
    char l = 'A';
    size_t size = 20;
    int pos = 0;
    str = (char*) malloc(size);
    
    while(l != '\0' && l != '\n')
    {
        scanf("%c", &l);

        str[pos] = l;
        pos++;

         if(pos > size)
        {
            str = (char*) realloc(str, size*2);
        }
    }

    fflush(stdin);
    
    return (char *)realloc(str, pos);
}

__host__ __device__ 
int shc(char* ca) 
{
    int h = 0, tam=0;

    while(ca[tam]!='\0')
        tam++;
      
    for (int i = 0; i < tam-1; i++) 
    {
        h = 31 * h + ca[i];
    }

    return h;
}

__device__ char * id2str(unsigned long long int n) 
{
    //Convert the id of the thread into the string in order to be checked
    int b=95, i, r, digit, p, count=0;
    char * a, * res;// clrscr();
    p=n;
 
    a = (char*) malloc(100); //cudaMallocManaged(&a, 100);

    do 
    {
        r=p%b;
        digit=32+r;
        a[count]=digit;
        count++;
        p=p/b;
    } 
    while(p!=0);

    res = (char*) malloc(count);

    for(i=count-1; i>=0; --i) res[count-i-1] = a[i];

    free(a);

    return res;
}

__global__ void findcollisions(int hash, int f)
{
    unsigned long long int id = blockDim.x * blockIdx.x + threadIdx.x; //+ i* /*4e40*/;
    
    char* trystr = id2str(id);

    int hc = shc(trystr);

    if(hc == hash)
    {
        printf("Collision found with string %s, hash code: %d\n", trystr, hc);
    }
    else
    {
        int tam = 0;

        while(trystr[tam]!='\0')
        {
            tam++;
        }
        
        char * reverse;
        reverse = (char *) malloc(tam); //cudaMallocManaged(&reverse, tam);
       
        for(int i=0;i<tam;i++)
        {
            reverse[i] = trystr[i];
        }

        if(trystr[tam-1] == ' ')
            for(int i=tam-1; i>=0; --i) reverse[tam-i-1] = trystr[i];

        hc = shc(reverse);

        if(hc == hash)
        {
            printf("Collision found with string %s, hash code: %d", trystr, hc);
        }

        free(reverse);
    }

    free(trystr);
   
}


int main(void)
{
    char* input_string;

    printf("Introduce una cadena: ");

   
    input_string = getstr(stdin, input_string);

    int hash = shc(input_string);
    
    findcollisions<<<1073741824,1024>>>(hash, 0); //<<<2^23, 2^10>>>

   
    hipDeviceSynchronize();
    
    free(input_string);

    return 0;
}