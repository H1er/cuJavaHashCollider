#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>


char * getstr(FILE* f, char * str)
{
    char l = 'A';
    size_t size = 20;
    int pos = 0;
    str = (char*) malloc(size);
    
    while(l != '\n')
    {
        scanf("%c", &l);

        str[pos] = l;
        pos++;

        if(pos > size)
            str = (char*) realloc(str, size^2);
    }

    fflush(stdin);
    
    str = (char *) realloc(str, pos);

    str[pos-1] = '\0';
    return str;
}

__device__ __host__  int shc(char* ca, int tam) 
{
    int h = 0;
    
    for (int i = 0; i < tam; i++) 
    {
        h = 31 * h + ca[i];
    }

    return h;
}

__device__ char * id2str(unsigned long long int n) 
{
    //Convert the id of the thread into the string in order to be checked
    int b=96, r, asciChar, count=0;
    char * a, * res;
 
    a = (char*) malloc(100);

    do 
    {
        r=n%b;
        asciChar=32+r;
        a[count]=asciChar;
        count++;
        n=n/b;
    } 
    while(n!=0);

    res = (char*) malloc(count+1);

    for(int i=count-1; i>=0; --i) res[count-i-1] = a[i];

    free(a);

    res[count] = '\0';

    return res;
}

__device__ void showProgress(char* trystr, int tam, unsigned long long int id)
{
    int n=0;

    for(int i=0;i<tam;i++)
    {
        if('!' != trystr[i]) 
        {
            n=1;
            break;
        }
    }

    if(id % 1000000000 == 0) printf("\n----- Id = %lld\n", id);

    if(n == 0) printf("\n***** Tam = %d -> ┤%s├\n", tam, trystr);
}

__global__ void findcollisions(int hash, int strLength)
{
    unsigned long long int id = (unsigned long long int) blockDim.x * (unsigned long long int) blockIdx.x + (unsigned long long int) threadIdx.x + ; //+ i* /*4e40*/;

    char *trystr = id2str(id);
    int hc;

    int tam = 0;
    
    while(trystr[tam]!='\0')
        tam++;

    showProgress(trystr, tam, id);

    hc = shc(trystr, tam);

    if(hc == hash) 
        printf("Collision found for string ┤%s├. Hashcode %d\n", trystr, hc);

    if(trystr[tam-1] == ' ')
    {
        char * reverse = (char *) malloc(tam+1);

        for(int i=tam-1; i>=0; --i)
            reverse[tam-i-1] = trystr[i];

        reverse[tam] = '\0';

        hc = shc(reverse, tam);
        
        if(hc == hash)
            printf("Collision found for string ┤%s├. Hashcode %d\n", reverse, hc);

        free(reverse);
    }
    free(trystr);

}

int main(void)
{
    char* input_string= NULL;

    printf("Introduce una cadena: ");
   
    input_string = getstr(stdin, input_string);

    int hash = shc(input_string, strlen(input_string));

    printf("\nSearching collisions for hashcode of ┤%s├: %d\n →→ START ←←\n\n", input_string, hash);

    findcollisions<<<pow(2,23),pow(2,10)>>>(hash, 0); //<<<2^23, 2^10>>>
   
    hipDeviceSynchronize();
    
    printf("\n →→ END ←←\n\n");

    free(input_string);

    return 0;
}