
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>


char * getstr(FILE* f, char * str)
{
    char l = 'A';
    size_t size = 20;
    int pos = 0;
    str = (char*) malloc(size);
    
    while(l != '\n')
    {
        scanf("%c", &l);

        str[pos] = l;
        pos++;

        if(pos > size)
            str = (char*) realloc(str, size^2);
    }

    fflush(stdin);
    
    str = (char *) realloc(str, pos);

    str[pos-1] = '\0';
    return str;
}

__device__ __host__  int shc(char* ca, int tam) 
{
    int h = 0;
    
    for (int i = 0; i < tam; i++) 
    {
        h = 31 * h + ca[i];
    }

    return h;
}

__device__ char * id2str(unsigned long long int n) 
{
    //Convert the id of the thread into the string in order to be checked
    int b=96, r, asciChar, count=0;
    char * a, * res;
 
    a = (char*) malloc(100);

    do 
    {
        r=n%b;
        asciChar=32+r;
        a[count]=asciChar;
        count++;
        n=n/b;
    } 
    while(n!=0);

    res = (char*) malloc(count+1);

    for(int i=count-1; i>=0; --i) res[count-i-1] = a[i];

    free(a);

    res[count] = '\0';

    return res;
}

__device__ void showProgress(char* trystr, int tam, unsigned long long int id)
{
    int n=0;

    for(int i=0;i<tam;i++)
    {
        if('!' != trystr[i]) 
        {
            n=1;
            break;
        }
    }

    if(id % 1000000000 == 0) printf("\n----- Id = %lld\n", id);

    if(n == 0) printf("\n***** %lld, Tam = %d -> ┤%s├\n", id, tam, trystr);
}

__global__ void findcollisions(int hash, unsigned long long int strLength)
{
    unsigned long long int id = blockDim.x * blockIdx.x + threadIdx.x; //+ i* /*4e40*/;

    char *trystr = id2str(id);
    int hc;

    int tam = 0;
    
    while(trystr[tam]!='\0')
        tam++;


    //if (oldId < 50) printf("%lld, ┤%s├\n", id, trystr);
    showProgress(trystr, tam, id);

    hc = shc(trystr, tam);

    if(hc == hash) 
        printf("Collision found for string ┤%s├. Hashcode %d\n", trystr, hc);

    if(trystr[tam-1] == ' ')
    {
        char * reverse = (char *) malloc(tam+1);

        for(int i=tam-1; i>=0; --i)
            reverse[tam-i-1] = trystr[i];

        reverse[tam] = '\0';

        hc = shc(reverse, tam);
        
        if(hc == hash)
            printf("Collision found for string ┤%s├. Hashcode %d\n", reverse, hc);

        free(reverse);
    }
    free(trystr);

}

int main(void)
{
    char* input_string= NULL;

    printf("Introduce una cadena: ");
   
    input_string = getstr(stdin, input_string);

    int length = strlen(input_string);

    int hash = shc(input_string, length);

    printf("\nSearching collisions for hashcode of ┤%s├: %d\n →→ START ←←\n\n", input_string, hash);

    findcollisions<<<pow(2,23),pow(2,10)>>>(hash, (unsigned long long int) pow(95, length)); //<<<2^23, 2^10>>>
   
    hipDeviceSynchronize();
    
    printf("\n →→ END ←←\n\n");

    free(input_string);

    return 0;
}